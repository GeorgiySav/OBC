#include "hip/hip_runtime.h"
#include "cuda_ops.h"

#include <cstdio>
#include <cstdlib>

#include <hipblas.h>
#include <hip/hip_runtime.h>

#include "cublas_utils.h"

namespace obc {
	namespace cuda {
		hipblasHandle_t cublasH = NULL;
		hipStream_t stream = NULL;

        void Init() {
			CUDA_CHECK(hipSetDevice(0));

			/* step 1: create cublas handle, bind a stream */
			CUBLAS_CHECK(hipblasCreate(&cublasH));

			CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
			CUBLAS_CHECK(hipblasSetStream(cublasH, stream));
		}

		double Dot(const std::vector<double>& A, const std::vector<double>& B) {
            double result = 0.0;

            double* d_A = nullptr;
            double* d_B = nullptr;

            /* step 2: copy data to device */
            CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_A), sizeof(double) * A.size()));
            CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_B), sizeof(double) * B.size()));

            CUDA_CHECK(hipMemcpyAsync(d_A, A.data(), sizeof(double) * A.size(), hipMemcpyHostToDevice,
                stream));
            CUDA_CHECK(hipMemcpyAsync(d_B, B.data(), sizeof(double) * B.size(), hipMemcpyHostToDevice,
                stream));

            /* step 3: compute */
            CUBLAS_CHECK(hipblasDdot(cublasH, A.size(), d_A, 1, d_B, 1, &result));

            CUDA_CHECK(hipStreamSynchronize(stream));

            /* free resources */
            CUDA_CHECK(hipFree(d_A));
            CUDA_CHECK(hipFree(d_B));

            return result;
		}

        void MatrixVecMul(const std::vector<double>& A, const int m, const int n, bool transpose,
			const std::vector<double>& x, 
			std::vector<double>& y) {

			const int lda = m;
			double alpha = 1.0;
			double beta = 0.0;

			double* d_A = nullptr;
			double* d_x = nullptr;
			double* d_y = nullptr;

			hipblasOperation_t transa = transpose ? HIPBLAS_OP_T : HIPBLAS_OP_N;

			/* step 2: copy data to device */
			CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_A), sizeof(double) * A.size()));
			CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_x), sizeof(double) * x.size()));
			CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_y), sizeof(double) * y.size()));

			CUDA_CHECK(hipMemcpyAsync(d_A, A.data(), sizeof(double) * A.size(), hipMemcpyHostToDevice,
                				stream));
			CUDA_CHECK(hipMemcpyAsync(d_x, x.data(), sizeof(double) * x.size(), hipMemcpyHostToDevice,
                				stream));

			/* step 3: compute */
			CUBLAS_CHECK(
				hipblasDgemv(cublasH, transa, m, n, &alpha, d_A, lda, d_x, 1, &beta, d_y, 1));

			CUDA_CHECK(hipMemcpyAsync(y.data(), d_y, sizeof(double) * y.size(), hipMemcpyDeviceToHost,
                				stream));

			CUDA_CHECK(hipStreamSynchronize(stream));

			/* free resources */
			CUDA_CHECK(hipFree(d_A));
			CUDA_CHECK(hipFree(d_x));
			CUDA_CHECK(hipFree(d_y));
        }

		void MatrixMatrixMul(
			const int m, const int k, const int n,
			const std::vector<double>& A, bool transposeA,
			const std::vector<double>& B, bool transposeB,
			std::vector<double>& C) {

			const int lda = transposeA ? k : m;
			const int ldb = transposeB ? n : k;
			const int ldc = m;

			double alpha = 1.0;
			double beta = 0.0;

			double* d_A = nullptr;
			double* d_B = nullptr;
			double* d_C = nullptr;

			hipblasOperation_t transa = transposeA ? HIPBLAS_OP_T : HIPBLAS_OP_N;
			hipblasOperation_t transb = transposeB ? HIPBLAS_OP_T : HIPBLAS_OP_N;

			CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_A), sizeof(double) * A.size()));
			CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_B), sizeof(double) * B.size()));
			CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_C), sizeof(double) * C.size()));

			CUDA_CHECK(hipMemcpyAsync(d_A, A.data(), sizeof(double) * A.size(), hipMemcpyHostToDevice,
				stream));
			CUDA_CHECK(hipMemcpyAsync(d_B, B.data(), sizeof(double) * B.size(), hipMemcpyHostToDevice,
				stream));

			CUBLAS_CHECK(
				hipblasDgemm(cublasH, transa, transb, m, n, k, &alpha, d_A, lda, d_B, ldb, &beta, d_C, ldc));

			CUDA_CHECK(hipMemcpyAsync(C.data(), d_C, sizeof(double) * C.size(), hipMemcpyDeviceToHost, stream));

			CUDA_CHECK(hipStreamSynchronize(stream));

			CUDA_CHECK(hipFree(d_A));
			CUDA_CHECK(hipFree(d_B));
			CUDA_CHECK(hipFree(d_C));
		}

		__global__ void VecVecAddKernel(const double* A, const double* B, double* C, int N) {
			// Get our global thread ID
			int id = blockIdx.x * blockDim.x + threadIdx.x;

			// Make sure we do not go out of bounds
			if (id < N)
				C[id] = A[id] + B[id];
		}

		void VecVecAdd(const std::vector<double>& A, const std::vector<double>& B, std::vector<double>& C) {
			const int N = A.size();
			const int size = N * sizeof(double);

			double* d_A = nullptr;
			double* d_B = nullptr;
			double* d_C = nullptr;

			CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_A), size));
			CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_B), size));
			CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_C), size));

			CUDA_CHECK(hipMemcpy(d_A, A.data(), size, hipMemcpyHostToDevice));
			CUDA_CHECK(hipMemcpy(d_B, B.data(), size, hipMemcpyHostToDevice));

			const int threads_per_block = 256;
			const int num_blocks = (N + threads_per_block - 1) / threads_per_block;

			VecVecAddKernel<<<num_blocks, threads_per_block>>>(d_A, d_B, d_C, N);
			hipDeviceSynchronize();

			CUDA_CHECK(hipMemcpy(C.data(), d_C, size, hipMemcpyDeviceToHost));

			CUDA_CHECK(hipFree(d_A));
			CUDA_CHECK(hipFree(d_B));
			CUDA_CHECK(hipFree(d_C));
		}

		__global__ void VecScalarAddKernel(const double* A, double scalar, double* y, int N) {
			// Get our global thread ID
			int id = blockIdx.x * blockDim.x + threadIdx.x;

			// Make sure we do not go out of bounds
			if (id < N) {
				y[id] = y[id] + (A[id] * scalar);
			}
		}

		void MatrixMatrixAdd(const std::vector<double>& A, double scalar, std::vector<double>& y) {
			const int N = A.size();
			const int size = N * sizeof(double);

			double* d_A = nullptr;
			double* d_y = nullptr;

			CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_A), size));
			CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_y), size));

			CUDA_CHECK(hipMemcpy(d_A, A.data(), size, hipMemcpyHostToDevice));
			CUDA_CHECK(hipMemcpy(d_y, y.data(), size, hipMemcpyHostToDevice));

			const int threads_per_block = 256;
			const int num_blocks = (N + threads_per_block - 1) / threads_per_block;

			VecScalarAddKernel<<<num_blocks, threads_per_block>>>(d_A, scalar, d_y, N);
			hipDeviceSynchronize();

			CUDA_CHECK(hipMemcpy(y.data(), d_y, size, hipMemcpyDeviceToHost));

			CUDA_CHECK(hipFree(d_A));
			CUDA_CHECK(hipFree(d_y));
		}

		__global__ void VecVecElementwiseMulKernel(const double* A, const double* B, double* C, int N) {
			// Get our global thread ID
			int id = blockIdx.x * blockDim.x + threadIdx.x;

			// Make sure we do not go out of bounds
			if (id < N) {
				C[id] = A[id] * B[id];
			}
		}

		void VecVecElementwiseMul(const std::vector<double>& A, const std::vector<double>& B, std::vector<double>& C) {
			const int N = A.size();
			const int size = N * sizeof(double);

			double* d_A = nullptr;
			double* d_B = nullptr;
			double* d_C = nullptr;

			CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_A), size));
			CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_B), size));
			CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_C), size));

			CUDA_CHECK(hipMemcpy(d_A, A.data(), size, hipMemcpyHostToDevice));
			CUDA_CHECK(hipMemcpy(d_B, B.data(), size, hipMemcpyHostToDevice));

			const int threads_per_block = 256;
			const int num_blocks = (N + threads_per_block - 1) / threads_per_block;

			VecVecElementwiseMulKernel<<<num_blocks, threads_per_block>>>(d_A, d_B, d_C, N);
			hipDeviceSynchronize();

			CUDA_CHECK(hipMemcpy(C.data(), d_C, size, hipMemcpyDeviceToHost));

			CUDA_CHECK(hipFree(d_A));
			CUDA_CHECK(hipFree(d_B));
			CUDA_CHECK(hipFree(d_C));
		}

		__global__ void ValidCrossCorrelateKernel(const double* A, int a_height, int a_width,
			const double* B, int b_height, int b_width, bool rot180,
			double* C, int c_height, int c_width) {

			int i = threadIdx.x + blockIdx.x * blockDim.x;
			int j = threadIdx.y + blockIdx.y * blockDim.y;

			int c_index = j * c_width + i;

			for (int x = 0; x < b_width; x++) {
				for (int y = 0; y < b_height; y++) {
					int a_index = (j + y) * a_width + (i + x);
					int b_index = y * b_width + x;
					if (rot180)	
						b_index = (b_height * b_width) - 1 - b_index;


					C[c_index] += A[a_index] * B[b_index];
				}
			}
		}

		__global__ void FullCrossCorrelateKernel(const double* A, int a_height, int a_width,
			const double* B, int b_height, int b_width, bool rot180,
			double* C, int c_height, int c_width) {

			int adj_i = threadIdx.x + blockIdx.x * blockDim.x;
			int adj_j = threadIdx.y + blockIdx.y * blockDim.y;

			int i = adj_i - (b_width - 1);
			int j = adj_j - (b_height - 1);

			int c_index = adj_j * c_width + adj_i;

			for (int x = 0; x < b_width; x++) {

				if (i + x < 0 || i + x >= a_width)
					continue;

				for (int y = 0; y < b_height; y++) {

					if (j + y < 0 || j + y >= a_height)
						continue;

					int a_index = (j + y) * a_width + (i + x);
					int b_index = y * b_width + x;
					if (rot180)
						b_index = (b_height * b_width) - 1 - b_index;

					C[c_index] += A[a_index] * B[b_index];
				}
			}
		}

		void CrossCorrelate(
			const std::vector<double>& A, int a_offset, int a_height, int a_width,
			const std::vector<double>& B, int b_offset, int b_height, int b_width, bool rot180,
			std::vector<double>& C, int c_offset, int c_height, int c_width,
			bool full) {

			double* d_A = nullptr;
			double* d_B = nullptr;
			double* d_C = nullptr;

			const int a_size = sizeof(double) * (a_height * a_width);
			CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_A), a_size));
			const int b_size = sizeof(double) * (b_height * b_width);
			CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_B), b_size));
			const int c_size = sizeof(double) * (c_height * c_width);
			CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_C), c_size));
		
			CUDA_CHECK(hipMemcpy(d_A, A.data() + a_offset, a_size, hipMemcpyHostToDevice));
			CUDA_CHECK(hipMemcpy(d_B, B.data() + b_offset, b_size, hipMemcpyHostToDevice));
			CUDA_CHECK(hipMemcpy(d_C, C.data() + c_offset, c_size, hipMemcpyHostToDevice));

			dim3 grid(1, 1, 1);
			dim3 block;

			if (full) {
				block = dim3(a_width - (1 - b_width), a_height - (1 - b_height), 1);
				FullCrossCorrelateKernel<<<grid, block>>>(d_A, a_height, a_width, d_B, b_height, b_width, rot180, d_C, c_height, c_width);
			}
			else {
				block = dim3(a_width - b_width + 1, a_height - b_height + 1, 1);
				ValidCrossCorrelateKernel<<<grid, block>>>(d_A, a_height, a_width, d_B, b_height, b_width, rot180, d_C, c_height, c_width);
			}

			hipDeviceSynchronize();

			CUDA_CHECK(hipMemcpy(C.data() + c_offset, d_C, c_size, hipMemcpyDeviceToHost));

			CUDA_CHECK(hipFree(d_A));
			CUDA_CHECK(hipFree(d_B));
			CUDA_CHECK(hipFree(d_C));
		}

		// template instantiation
		template void ApplyFunc<FunctionType::kSigmoid>(std::vector<double>& A);
		template void ApplyFunc<FunctionType::kSigmoid>(const std::vector<double>& A, std::vector<double>& y);
		template void ApplyFunc<FunctionType::kSigmoidPrime>(std::vector<double>& A);
		template void ApplyFunc<FunctionType::kSigmoidPrime>(const std::vector<double>& A, std::vector<double>& y);

		template void ApplyFunc<FunctionType::kReLu>(std::vector<double>& A);
		template void ApplyFunc<FunctionType::kReLu>(const std::vector<double>& A, std::vector<double>& y);
		template void ApplyFunc<FunctionType::kReLuPrime>(std::vector<double>& A);
		template void ApplyFunc<FunctionType::kReLuPrime>(const std::vector<double>& A, std::vector<double>& y);

		__device__ double Sigmoid(double x) {
			return 1 / (1 + exp(-x));
		}
		__device__ double SigmoidPrime(double x) {
			double s = Sigmoid(x);
			return s * (1 - s);
		}
		__device__ double ReLu(double x) {
			return x > 0 ? x : 0;
		}
		__device__ double ReLuPrime(double x) {
			return x > 0 ? 1 : 0;
		}
		template <FunctionType func>
		__global__ void ApplyFuncKernel(double* A, int N) {
			// Get our global thread ID
			int id = blockIdx.x * blockDim.x + threadIdx.x;

			// Make sure we do not go out of bounds
			if (id < N) {
				if constexpr (func == FunctionType::kSigmoid) {
					A[id] = Sigmoid(A[id]);
				}
				else if constexpr (func == FunctionType::kSigmoidPrime) {
					A[id] = SigmoidPrime(A[id]);
				}
				else if constexpr (func == FunctionType::kReLu) {
					A[id] = ReLu(A[id]);
				}	
				else if constexpr (func == FunctionType::kReLuPrime) {
					A[id] = ReLuPrime(A[id]);
				}
			}
		}
		template <FunctionType func>
		void ApplyFunc(std::vector<double>& A) {
			const int N = A.size();
			const int size = N * sizeof(double);

			double* d_A = nullptr;

			CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_A), size));

			CUDA_CHECK(hipMemcpy(d_A, A.data(), size, hipMemcpyHostToDevice));

			const int threads_per_block = 256;
			const int num_blocks = (N + threads_per_block - 1) / threads_per_block;

			ApplyFuncKernel<func><<<num_blocks, threads_per_block>>>(d_A, N);
			hipDeviceSynchronize();

			CUDA_CHECK(hipMemcpy(A.data(), d_A, size, hipMemcpyDeviceToHost));

			CUDA_CHECK(hipFree(d_A));
		}
		template <FunctionType func>
		__global__ void ApplyFuncKernel(const double* A, double* y, int N) {
			// Get our global thread ID
			int id = blockIdx.x * blockDim.x + threadIdx.x;

			// Make sure we do not go out of bounds
			if (id < N) {
				if constexpr (func == FunctionType::kSigmoid) {
					y[id] = Sigmoid(A[id]);
				}
				else if constexpr (func == FunctionType::kSigmoidPrime) {
					y[id] = SigmoidPrime(A[id]);
				}
				else if constexpr (func == FunctionType::kReLu) {
					y[id] = ReLu(A[id]);
				}
				else if constexpr (func == FunctionType::kReLuPrime) {
					y[id] = ReLuPrime(A[id]);
				}
			}
		}
		template <FunctionType func>
		void ApplyFunc(const std::vector<double>& A, std::vector<double>& y) {
			const int N = A.size();
			const int size = N * sizeof(double);

			double* d_A = nullptr;
			double* d_y = nullptr;

			CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_A), size));
			CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_y), size));

			CUDA_CHECK(hipMemcpy(d_A, A.data(), size, hipMemcpyHostToDevice));

			const int threads_per_block = 256;
			const int num_blocks = (N + threads_per_block - 1) / threads_per_block;

			ApplyFuncKernel<func><<<num_blocks, threads_per_block>>>(d_A, d_y, N);
			hipDeviceSynchronize();

			CUDA_CHECK(hipMemcpy(y.data(), d_y, size, hipMemcpyDeviceToHost));

			CUDA_CHECK(hipFree(d_A));
			CUDA_CHECK(hipFree(d_y));
		}

        void Shutdown() {
            CUBLAS_CHECK(hipblasDestroy(cublasH));
            CUDA_CHECK(hipStreamDestroy(stream));
            CUDA_CHECK(hipDeviceReset());
        }
	}
}
